
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <vector>
#include <chrono>
#include <string>
//
#include <hip/hip_complex.h>
//
#define MaxIteration 255;  //!< Le nombre max d'itération est 255, soit de base le blanc.
//
static void HandleError(	hipError_t err,
                            const char *file,
                            int line )
{
    if (err != hipSuccess)
    {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
        file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
// Cette méthode sert uniquement à sauvegarder le vecteur sous forme d'une image en niveau de gris sur 8 bits.
void save_pgm(  const char*                         filename,
                const size_t                        width,
                const size_t                        height,
                const std::vector<std::uint8_t>&   data)
{
    std::ofstream fout{ filename };
    // L'en-tête
    fout << "P2\n" << width << " " << height << " 255\n";
    for (size_t row = 0; row < height; ++row)
    {
        for (size_t col = 0; col < width; ++col)
        {
            fout << (col ? " " : "")
                 << static_cast<unsigned>(data[row * width + col]);
        }
        fout << "\n";
    }
    fout.close();
}
//
__global__ void mandel_kernel_double(/* TODO */)
{
    // TODO : garder le même algorithme qu'en C++, il faut juste "traduire" les éléments
    // de C++ en CUDA.
    // Que devient la boucle 'for' si on souhaite calculer chaque pixel dans un thread ?
    // Attention à l'indice global du thread, il faut vérifier que nous sommes bien dans les bornes de l'image !
    // Utiliser les méthodes et les éléments fournis dans le PDF concernant les nombres complexes avec CUDA 
    // (on peut aussi explorer 'cuComplex.h' pour voir ce qui s'y trame).
}
//
int main(int argc, char* argv[])
{
    if (argc != 3)
    {
        std::cerr << "Usage:\n"
                  << argv[0] << " [width] [height]\n";
        return 1;
    }
    const size_t width  = std::stoul(argv[1]);
    const size_t height = std::stoul(argv[2]);
    std::vector<std::uint8_t> image(height * width, 0);
    // Note : il est possible de manipuler le pointeur de données sous-jacent au vecteur via la méthode '.data()'
    auto t0 = std::chrono::high_resolution_clock::now();
    // TODO : Appeler mandel_kernel_double
    auto t1 = std::chrono::high_resolution_clock::now();
    std::cout << "Generation of Mandelbrot set for image size " << width << " x " << height << " took "
              << std::chrono::duration<double>(t1-t0).count() << " seconds (GPU version)\n";
    save_pgm("output_GPU.pgm", width, height, image);
}
